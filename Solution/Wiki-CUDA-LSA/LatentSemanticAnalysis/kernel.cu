#include "hip/hip_runtime.h"

#include<iostream>
#include<map>
#include<tfidf.cuh>
#include<svd.cuh>
#include"lsa.cuh"

const auto test_TD_docs = 12;
const auto test_TD_terms = 9;

static double test_doc_term_freq[test_TD_docs * test_TD_terms] = {
	1, 0, 0, 1, 0, 0, 0, 0, 0,
	1, 0, 1, 0, 0, 0, 0, 0, 0,
	1, 1, 0, 0, 0, 0, 0, 0, 0,
	0, 1, 1, 0, 1, 0, 0, 0, 0,
	0, 1, 1, 2, 0, 0, 0, 0, 0,
	0, 1, 0, 0, 1, 0, 0, 0, 0,
	0, 1, 0, 0, 1, 0, 0, 0, 0,
	0, 0, 1, 1, 0, 0, 0, 0, 0,
	0, 1, 0, 0, 0, 0, 0, 0, 1,
	0, 0, 0, 0, 0, 1, 1, 1, 0,
	0, 0, 0, 0, 0, 0, 1, 1, 1,
	0, 0, 0, 0, 0, 0, 0, 1, 1,
};

const auto test_k = 2;

const auto numConcept = 2;
const auto numTerms = 2;
const auto numDocs = 2;

void printColMajorMatrix(int m, int n, const double*A, int lda, const char* name) {
	for (int row = 0; row < m; row++) {
		for (int col = 0; col < n; col++) {
			double Areg = A[col * m + row];
			printf("%s(%d,%d) = %f\n", name, row, col, Areg);
		}
	}
}

void printRowMajorMatrix(int m, int n, const double*A, const char* name) {
	for (int row = 0; row < m; row++) {
		for (int col = 0; col < n; col++) {
			double Areg = A[row * n + col];
			printf("%s(%d,%d) = %f\n", name, row, col, Areg);
		}
	}
}

void printConcepts(const int numConcept, const int numTerms, const int numDocs, const std::vector<std::vector<std::tuple<int, double>>> topTerms, const std::vector<std::vector<std::tuple<int, double>>> topDocs) {
	for (int c = 0; c < numConcept; c++) {
		std::cout << "Concept " << c << ":" << std::endl;
		std::cout << "\t" << "Top Term Idx: ";
		for (int i = 0; i < numTerms; i++) {
			std::cout << std::get<0>(topTerms[c][i]) << "(" << std::get<1>(topTerms[c][i]) << "), ";
		}
		std::cout << std::endl;
		std::cout << "\t" << "Top Doc Idx: ";
		for (int i = 0; i < numDocs; i++) {
			std::cout << std::get<0>(topDocs[c][i]) << "(" << std::get<1>(topDocs[c][i]) << "), ";
		}
		std::cout << std::endl;
	}
}

void printTermTerm(const int lookUpIdx, const std::vector<std::tuple<int, double>> termTerm) {
	std::cout << "Term correlation to term index " << lookUpIdx << " ranking:" << std::endl;
	std::cout << "\t";
	for(auto & t : termTerm) {
		std::cout << std::get<0>(t) << "(" << std::get<1>(t) << "), ";
	}
	std::cout << std::endl;
}

int main(){
	const int m = test_TD_docs;
	const int n = test_TD_terms;
	const int k = test_k;
	const int lda = m;
	//TD/IDF
	tfidf(m, n, test_doc_term_freq);
	//svd
	double U[lda * k];
	double S[k];
	double VT[k * n];
	printf("A = \n");
	printColMajorMatrix(m, n, test_doc_term_freq, lda, "A");
	printf("=====\n");
	approximate_svd(m, n, k, test_doc_term_freq, U, S, VT);
	printf("U = \n");
	printRowMajorMatrix(m, k, U, "U");
	printf("=====\n");
	printf("S = \n");
	printRowMajorMatrix(k, 1, S, "S");
	printf("=====\n");
	printf("VT = \n");
	printRowMajorMatrix(k, n, VT, "VT");
	printf("=====\n");
	printf("V = \n");
	double V[n * k];
	memcpy(V, VT, sizeof(double) * k * n);
	transpose(k, n, V);
	printRowMajorMatrix(n, k, V, "V");
	printf("=====\n");
	//concepts
	auto topTerms = topTermsInTopConcepts(k, n, VT, numConcept, numTerms);
	auto topDocs = topDocsInTopConcepts(m, k, U, numConcept, numDocs);
	printConcepts(numConcept, numTerms, numDocs, topTerms, topDocs);
	printf("=====\n");
	//corrolated
	printf("V*S = \n");
	double VS[n * k];
	memcpy(VS, V, sizeof(double) * k * n);
	multiplyByDiagonalMatrix(n, k, VS, S);
	printRowMajorMatrix(n, k, VS, "VS");
	printf("=====\n");

	printf("norm(V*S) = \n");
	double normVS[n * k];
	memcpy(normVS, VS, sizeof(double) * k * n);
	rowsNormalized(n, k, normVS);
	printRowMajorMatrix(n, k, normVS, "normVS");
	printf("=====\n");

	printf("U*S = \n");
	double US[m * k];
	memcpy(US, U, sizeof(double) * k * m);
	multiplyByDiagonalMatrix(m, k, US, S);
	printRowMajorMatrix(m, k, US, "US");
	printf("=====\n");

	printf("norm(U*S) = \n");
	double normUS[m * k];
	memcpy(normUS, US, sizeof(double) * k * m);
	rowsNormalized(m, k, normUS);
	printRowMajorMatrix(m, k, normUS, "normUS");
	printf("=====\n");

	int lookUpTermIdx = 0;
	auto termTerm = topTermsForTerm(n, k, normVS, lookUpTermIdx);
	printTermTerm(lookUpTermIdx, termTerm);

    return 0;
}