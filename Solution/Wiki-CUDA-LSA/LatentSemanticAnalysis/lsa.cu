#include "hip/hip_runtime.h"
#include"lsa.cuh"
#include<svd.cuh>

#include<cassert>
#include<algorithm>


//TODO: return void
//TODO: parallelize
std::vector<std::vector<std::tuple<int, double>>> topTermsInTopConcepts(int rows, int cols, double * matrix_VT, int numConcepts, int numTerms) {
	assert(numConcepts <= rows && numTerms <= cols);
	auto topTerms = std::vector<std::vector<std::tuple<int, double>>>();
	// VT: concept->row, term->col
	for (int i = 0; i < numConcepts; i++) {
		auto termWeights = std::vector<std::tuple<int, double>>();
		for (int j = 0; j < cols; j++) {
			termWeights.push_back(std::make_tuple(j, matrix_VT[i * cols + j]));//<termIdx, termWeight>
		}
		sort(termWeights.begin(), termWeights.end(), [](auto a, auto b) {return std::get<1>(a) > std::get<1>(b); });//weight desc
		auto conceptTopTerms = std::vector<std::tuple<int, double>>();
		for (int j = 0; j < numTerms; j++) {
			conceptTopTerms.push_back(termWeights[j]);
		}
		topTerms.push_back(conceptTopTerms);
	}
	return topTerms;
}

//TODO: return void
//TODO: parallelize
std::vector<std::vector<std::tuple<int, double>>> topDocsInTopConcepts(int rows, int cols, double * matrix_U, int numConcepts, int numDocs) {
	assert(numConcepts <= cols && numDocs <= rows);
	auto topDocs = std::vector<std::vector<std::tuple<int, double>>>();
	//U: doc->row, concept->col
	for (int j = 0; j < numConcepts; j++) {
		auto docWeights = std::vector<std::tuple<int, double>>();
		for (int i = 0; i < rows; i++) {
			docWeights.push_back(std::make_tuple(i, matrix_U[i * cols + j]));
		}
		sort(docWeights.begin(), docWeights.end(), [](auto a, auto b) {return std::get<1>(a) > std::get<1>(b); });//weight desc
		auto conceptTopDocs = std::vector<std::tuple<int, double>>();
		for (int i = 0; i < numDocs; i++) {
			conceptTopDocs.push_back(docWeights[i]);
		}
		topDocs.push_back(conceptTopDocs);
	}
	return topDocs;
}


//TODO: parallelize
void rowsNormalized(int rows, int cols, double * matrix) {
	for (int i = 0; i < rows; i++) {
		double sqrSum = 0;
		for (int j = 0; j < cols; j++) {
			sqrSum += pow(matrix[i * cols + j], 2);
		}
		for (int j = 0; j < cols; j++) {
			matrix[i * cols + j] /= sqrSum;
		}
	}
}

//TODO: return void
//TODO: parallelize
std::vector<std::tuple<int, double>> topTermsForTerm(int rows, int cols, double * normVS, int termIdx) {
	double * rowVec = new double[cols];
	memcpy(rowVec, normVS + termIdx * cols, sizeof(double) * cols);
	double * resultVec = new double[rows];
	multiply(rows, cols, 1, normVS, rowVec, resultVec);
	auto termScores = std::vector<std::tuple<int, double>>();
	for (int i = 0; i < rows; i++) {
		termScores.push_back(std::make_tuple(i, resultVec[i]));
	}
	sort(termScores.begin(), termScores.end(), [](auto a, auto b) {return std::get<1>(a) > std::get<1>(b); });//weight desc
	delete[] resultVec;
	delete[] rowVec;
	return termScores;
}