#include "hip/hip_runtime.h"
#include"lsa.cuh"
#include<svd.cuh>

#include<cassert>
#include<algorithm>

//TODO: return void
//TODO: parallelize
std::vector<std::vector<std::tuple<int, float>>> topElementsInTopConcepts(int rows, int cols, float * matrix, int numConcepts, int numElement) {
	assert(numConcepts <= cols && numElement <= rows);
	auto topElements = std::vector<std::vector<std::tuple<int, float>>>();
	//U: doc->row, concept->col
	for (int j = 0; j < numConcepts; j++) {
		auto elementWeights = std::vector<std::tuple<int, float>>();
		for (int i = 0; i < rows; i++) {
			elementWeights.push_back(std::make_tuple(i, matrix[i * cols + j]));
		}
		sort(elementWeights.begin(), elementWeights.end(), [](std::tuple<int, float> a, std::tuple<int, float> b) {return std::get<1>(a) > std::get<1>(b); });//weight desc
		auto conceptTopElements = std::vector<std::tuple<int, float>>();
		for (int i = 0; i < numElement; i++) {
			conceptTopElements.push_back(elementWeights[i]);
		}
		topElements.push_back(conceptTopElements);
	}
	return topElements;
}


//TODO: parallelize
void rowsNormalized(int rows, int cols, float * matrix) {
	for (int i = 0; i < rows; i++) {
		float sqrSum = 0;
		for (int j = 0; j < cols; j++) {
			sqrSum += pow(matrix[i * cols + j], 2);
		}
		for (int j = 0; j < cols; j++) {
			matrix[i * cols + j] /= sqrSum;
		}
	}
}

//TODO: return void
//TODO: parallelize
std::vector<std::tuple<int, float>> topsForTerm(int rows, int cols, float * normXS, int vRows, int vCols, float * normVS, int termIdx) {
	assert(cols == vCols);
	float * rowVec = new float[vCols];
	memcpy(rowVec, normVS + termIdx * vCols, sizeof(float) * vCols);
	float * resultVec = new float[rows];
	multiply(rows, cols, 1, normXS, rowVec, resultVec);
	auto docScores = std::vector<std::tuple<int, float>>();
	for (int i = 0; i < rows; i++) {
		docScores.push_back(std::make_tuple(i, resultVec[i]));
	}
	sort(docScores.begin(), docScores.end(), [](std::tuple<int, float> a, std::tuple<int, float> b) {return std::get<1>(a) > std::get<1>(b); });//weight desc
	delete[] resultVec;
	delete[] rowVec;
	return docScores;
}