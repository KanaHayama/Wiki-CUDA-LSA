#include "hip/hip_runtime.h"
#include"lsa.cuh"
#include<svd.cuh>

#include<cassert>
#include<algorithm>


//TODO: return void
//TODO: parallelize
std::vector<std::vector<std::tuple<int, float>>> topTermsInTopConcepts(int rows, int cols, float * matrix_VT, int numConcepts, int numTerms) {
	assert(numConcepts <= rows && numTerms <= cols);
	auto topTerms = std::vector<std::vector<std::tuple<int, float>>>();
	// VT: concept->row, term->col
	for (int i = 0; i < numConcepts; i++) {
		auto termWeights = std::vector<std::tuple<int, float>>();
		for (int j = 0; j < cols; j++) {
			termWeights.push_back(std::make_tuple(j, matrix_VT[i * cols + j]));//<termIdx, termWeight>
		}
		sort(termWeights.begin(), termWeights.end(), [](auto a, auto b) {return std::get<1>(a) > std::get<1>(b); });//weight desc
		auto conceptTopTerms = std::vector<std::tuple<int, float>>();
		for (int j = 0; j < numTerms; j++) {
			conceptTopTerms.push_back(termWeights[j]);
		}
		topTerms.push_back(conceptTopTerms);
	}
	return topTerms;
}

//TODO: return void
//TODO: parallelize
std::vector<std::vector<std::tuple<int, float>>> topDocsInTopConcepts(int rows, int cols, float * matrix_U, int numConcepts, int numDocs) {
	assert(numConcepts <= cols && numDocs <= rows);
	auto topDocs = std::vector<std::vector<std::tuple<int, float>>>();
	//U: doc->row, concept->col
	for (int j = 0; j < numConcepts; j++) {
		auto docWeights = std::vector<std::tuple<int, float>>();
		for (int i = 0; i < rows; i++) {
			docWeights.push_back(std::make_tuple(i, matrix_U[i * cols + j]));
		}
		sort(docWeights.begin(), docWeights.end(), [](auto a, auto b) {return std::get<1>(a) > std::get<1>(b); });//weight desc
		auto conceptTopDocs = std::vector<std::tuple<int, float>>();
		for (int i = 0; i < numDocs; i++) {
			conceptTopDocs.push_back(docWeights[i]);
		}
		topDocs.push_back(conceptTopDocs);
	}
	return topDocs;
}


//TODO: parallelize
void rowsNormalized(int rows, int cols, float * matrix) {
	for (int i = 0; i < rows; i++) {
		float sqrSum = 0;
		for (int j = 0; j < cols; j++) {
			sqrSum += pow(matrix[i * cols + j], 2);
		}
		for (int j = 0; j < cols; j++) {
			matrix[i * cols + j] /= sqrSum;
		}
	}
}

//TODO: return void
//TODO: parallelize
std::vector<std::tuple<int, float>> topTermsForTerm(int rows, int cols, float * normVS, int termIdx) {
	float * rowVec = new float[cols];
	memcpy(rowVec, normVS + termIdx * cols, sizeof(float) * cols);
	float * resultVec = new float[rows];
	multiply(rows, cols, 1, normVS, rowVec, resultVec);
	auto termScores = std::vector<std::tuple<int, float>>();
	for (int i = 0; i < rows; i++) {
		termScores.push_back(std::make_tuple(i, resultVec[i]));
	}
	sort(termScores.begin(), termScores.end(), [](auto a, auto b) {return std::get<1>(a) > std::get<1>(b); });//weight desc
	delete[] resultVec;
	delete[] rowVec;
	return termScores;
}