#include "hip/hip_runtime.h"
#include"tfidf.cuh"

#include<cmath>

//TODO: parallelize
void tfidf(int rows, int cols, double * matrix) {
	int numDocs = rows;
	int numTerms = cols;
	for (int termIdx = 0; termIdx < numTerms; termIdx++) {
		int includeDocs = 0;
		for (int docIdx = 0; docIdx < numDocs; docIdx++) {
			if (matrix[docIdx * cols + termIdx] > 0) {
				includeDocs++;
			}
		}
		double termIdf = numDocs / (log10(includeDocs) + 1);
		for (int docIdx = 0; docIdx < numDocs; docIdx++) {
			matrix[docIdx * cols + termIdx] *= termIdf;
		}
	}
}