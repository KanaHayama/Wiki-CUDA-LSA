/*
 * How to compile (assume cuda is installed at /usr/local/cuda/)
 *   nvcc -c -I/usr/local/cuda/include svd_example.cpp
 *   g++ -o a.out svd_example.o -L/usr/local/cuda/lib64 -lcudart -lcublas -lcusolver
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "svd.cuh"

void printMatrix(int m, int n, const double*A, int lda, const char* name) {
	for (int row = 0; row < m; row++) {
		for (int col = 0; col < n; col++) {
			double Areg = A[row * n + col];
			printf("%s(%d,%d) = %f\n", name, row, col, Areg);
		}
	}
}

void test_precise() {
	const int m = 3;
	const int n = 2;
	const int lda = m;
	/*       | 1 4  |
	 *   A = | 2 2  |
	 *       | 5 1  |
	 */
	double A[lda*n] = { 1.0, 4.0, 2.0, 2.0, 5.0, 1.0 };
	double result_A[lda*n];
	double U[lda*m]; // m-by-m unitary matrix 
	double VT[lda*n];  // n-by-n unitary matrix
	double S[n]; // singular value

	printf("A = (matlab base-1)\n");
	printMatrix(m, n, A, lda, "A");
	printf("=====\n");

	svd(m, n, A, U, S, VT);

	printf("=====\n");

	printf("S = (matlab base-1)\n");
	printMatrix(n, 1, S, lda, "S");
	printf("=====\n");

	printf("U = (matlab base-1)\n");
	printMatrix(m, m, U, lda, "U");
	printf("=====\n");

	printf("VT = (matlab base-1)\n");
	printMatrix(n, n, VT, lda, "VT");
	printf("=====\n");

	svd_r(m, n, U, S, VT, result_A);
	printMatrix(m, n, result_A, lda, "rsult A");
}

void test_approximate() {
	const int m = 3;
	const int n = 2;
	const int lda = m;
	/*       | 1 4  |
	 *   A = | 2 2  |
	 *       | 5 1  |
	 */
	const int k = 2;
	double A[lda*n] = { 1.0, 4.0, 2.0, 2.0, 5.0, 1.0 };
	double result_A[lda*n];
	double U[lda * k]; // m-by-m unitary matrix 
	double S[k]; // singular value
	double VT[k * n];  // n-by-n unitary matrix	

	printf("A = (matlab base-1)\n");
	printMatrix(m, n, A, lda, "A");
	printf("=====\n");

	aproximate_svd(m, n, k, A, U, S, VT);

	printf("=====\n");

	printf("S = (matlab base-1)\n");
	printMatrix(k, 1, S, lda, "S");
	printf("=====\n");

	printf("U = (matlab base-1)\n");
	printMatrix(m, k, U, lda, "U");
	printf("=====\n");

	printf("VT = (matlab base-1)\n");
	printMatrix(k, n, VT, lda, "VT");
	printf("=====\n");
}

int main(int argc, char*argv[]) {
	test_precise();
	test_approximate();

	return 0;
}