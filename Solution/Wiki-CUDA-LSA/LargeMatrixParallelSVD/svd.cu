#include"svd.cuh"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>

void transpose_device(int rows, int cols, float * d_Matrix) {
	if (rows == 1 || cols == 1) {
		return;
	}
	hipError_t cudaStat = hipSuccess;
	float * d_Result = NULL;
	cudaStat = hipMalloc((void**)&d_Result, sizeof(float) * rows * cols);
	assert(hipSuccess == cudaStat);
	const float alpha = 1;
	const float beta = 0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
	hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, rows, cols, &alpha, d_Matrix, cols, &beta, d_Matrix, rows, d_Result, rows);
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
	cudaStat = hipMemcpy(d_Matrix, d_Result, sizeof(float) * rows * cols, hipMemcpyDeviceToDevice);
	assert(hipSuccess == cudaStat);
	if (d_Result) hipFree(d_Result);
	hipblasDestroy(handle);
}

void transpose(int rows, int cols, float * matrix) {
	if (rows == 1 || cols == 1) {
		return;
	}
	hipError_t cudaStat = hipSuccess;
	float * d_Matrix = NULL;
	cudaStat = hipMalloc((void**)&d_Matrix, sizeof(float) * rows * cols);
	assert(hipSuccess == cudaStat);
	cudaStat = hipMemcpy(d_Matrix, matrix, sizeof(float) * rows * cols, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);
	transpose_device(rows, cols, d_Matrix);
	cudaStat = hipMemcpy(matrix, d_Matrix, sizeof(float) * rows * cols, hipMemcpyDeviceToHost);
	assert(hipSuccess == cudaStat);
	if (d_Matrix) hipFree(d_Matrix);
}

inline void row_major_2_col_major_device(int rows, int cols, float * matrix) {
	transpose_device(rows, cols, matrix);
}

inline void col_major_2_row_major_device(int rows, int cols, float * matrix) {
	transpose_device(cols, rows, matrix);
}

void svd(int rows, int cols, float * matrix_A, float * matrix_U, float * array_S, float * matrix_VT) {
	assert(rows >= cols);
	// step 1: create cusolverDn handle
	hipsolverHandle_t cusolverH = NULL;
	hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
	cusolver_status = hipsolverDnCreate(&cusolverH);
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

	// step 2: copy A and B to device
	const int lda = rows;
	hipError_t cudaStat = hipSuccess;
	float * d_A = NULL;
	cudaStat = hipMalloc((void**)&d_A, sizeof(float) * lda * cols);
	assert(hipSuccess == cudaStat);
	float *d_U = NULL;
	cudaStat = hipMalloc((void**)&d_U, sizeof(float) * lda * rows);
	assert(hipSuccess == cudaStat);
	float *d_S = NULL;
	cudaStat = hipMalloc((void**)&d_S, sizeof(float) * cols);
	assert(hipSuccess == cudaStat);
	float *d_VT = NULL;
	cudaStat = hipMalloc((void**)&d_VT, sizeof(float) * lda * cols);
	assert(hipSuccess == cudaStat);
	int * devInfo = NULL;
	cudaStat = hipMalloc((void**)&devInfo, sizeof(int));
	assert(hipSuccess == cudaStat);

	cudaStat = hipMemcpy(d_A, matrix_A, sizeof(float) * lda * cols, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);

	row_major_2_col_major_device(lda, cols, d_A);

	// step 3: query working space of SVD
	int lwork = 0;
	cusolver_status = hipsolverDnDgesvd_bufferSize(
		cusolverH,
		rows,
		cols,
		&lwork);
	assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);

	float *d_work = NULL;
	cudaStat = hipMalloc((void**)&d_work, sizeof(float) * lwork);
	assert(hipSuccess == cudaStat);

	// step 4: compute SVD
	signed char jobu = 'A'; // all m columns of U
	signed char jobvt = 'A'; // all n columns of VT
	float *d_rwork = NULL;
	cusolver_status = hipsolverDnSgesvd(
		cusolverH,
		jobu,
		jobvt,
		rows,
		cols,
		d_A,
		lda,
		d_S,
		d_U,
		lda,  // ldu
		d_VT,
		lda, // ldvt,
		d_work,
		lwork,
		d_rwork,
		devInfo);
	cudaStat = hipDeviceSynchronize();
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
	assert(hipSuccess == cudaStat);

	// step 5: copy back
	if (matrix_U) {
		col_major_2_row_major_device(lda, rows, d_U);
		cudaStat = hipMemcpy(matrix_U, d_U, sizeof(float) * lda * rows, hipMemcpyDeviceToHost);
		assert(hipSuccess == cudaStat);
	}
	if (array_S) {
		cudaStat = hipMemcpy(array_S, d_S, sizeof(float) * cols, hipMemcpyDeviceToHost);
		assert(hipSuccess == cudaStat);
	}
	if (matrix_VT) {
		col_major_2_row_major_device(lda, cols, d_VT);
		cudaStat = hipMemcpy(matrix_VT, d_VT, sizeof(float) * lda * cols, hipMemcpyDeviceToHost);
		assert(hipSuccess == cudaStat);
	}
	int info_gpu = 0;
	cudaStat = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
	assert(hipSuccess == cudaStat);
	assert(0 == info_gpu);

	// step 6: free
	if (d_A) hipFree(d_A);
	if (d_S) hipFree(d_S);
	if (d_U) hipFree(d_U);
	if (d_VT) hipFree(d_VT);
	if (devInfo) hipFree(devInfo);
	if (d_work) hipFree(d_work);
	if (d_rwork) hipFree(d_rwork);
	if (cusolverH) hipsolverDnDestroy(cusolverH);

	hipDeviceReset();
}

void svd_r(int rows, int cols, float * matrix_U, float * array_S, float * matrix_VT, float * matrix_A) {
	// copy
	const int lda = rows;
	hipError_t cudaStat = hipSuccess;
	float *d_U = NULL;
	cudaStat = hipMalloc((void**)&d_U, sizeof(float) * lda * rows);
	assert(hipSuccess == cudaStat);
	float *d_S = NULL;
	cudaStat = hipMalloc((void**)&d_S, sizeof(float) * cols);
	assert(hipSuccess == cudaStat);
	float *d_VT = NULL;
	cudaStat = hipMalloc((void**)&d_VT, sizeof(float) * lda * cols);
	assert(hipSuccess == cudaStat);
	float * d_W = NULL;  // W = S*VT
	cudaStat = hipMalloc((void**)&d_W, sizeof(float) * lda * cols);
	assert(hipSuccess == cudaStat);
	float * d_A = NULL;
	cudaStat = hipMalloc((void**)&d_A, sizeof(float) * lda * cols);
	assert(hipSuccess == cudaStat);
	int * devInfo = NULL;
	cudaStat = hipMalloc((void**)&devInfo, sizeof(int));
	assert(hipSuccess == cudaStat);

	cudaStat = hipMemcpy(d_U, matrix_U, sizeof(float) * lda * rows, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);
	cudaStat = hipMemcpy(d_S, array_S, sizeof(float) * cols, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);
	cudaStat = hipMemcpy(d_VT, matrix_VT, sizeof(float) * lda * cols, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);

	// create cublas handle
	hipblasHandle_t cublasH = NULL;
	hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
	cublas_status = hipblasCreate(&cublasH);
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	// W = S*VT
	cublas_status = hipblasSdgmm(
		cublasH,
		HIPBLAS_SIDE_LEFT,
		cols,
		cols,
		d_VT,
		lda,
		d_S,
		1,
		d_W,
		lda);
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	// A := U*W
	assert(hipSuccess == cudaStat);
	const float h_one = 1;
	const float h_zero = 0;
	cublas_status = hipblasSgemm(
		cublasH,
		HIPBLAS_OP_N, // U
		HIPBLAS_OP_N, // W
		rows, // number of rows of A
		cols, // number of columns of A
		cols, // number of columns of U 
		&h_one, /* host pointer */
		d_U, // U
		lda,
		d_W, // W
		lda,
		&h_zero, /* hostpointer */
		d_A,
		lda);
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	// copy
	if (matrix_A) {
		cudaStat = hipMemcpy(matrix_A, d_A, sizeof(float) * lda * cols, hipMemcpyDeviceToHost);
		assert(hipSuccess == cudaStat);
	}

	// free
	if (d_A) hipFree(d_A);
	if (d_S) hipFree(d_S);
	if (d_U) hipFree(d_U);
	if (d_VT) hipFree(d_VT);
	if (devInfo) hipFree(devInfo);
	if (cublasH) hipblasDestroy(cublasH);

	hipDeviceReset();
}

//TODO: use k to speed up svd, rather than apply after svd
void approximate_svd(int rows, int cols, int k, float * matrix_A, float * matrix_U, float * array_S, float * matrix_VT) {
	assert(rows >= cols);
	assert(k <= rows && k <= cols);
	// step 1: create cusolverDn handle
	hipsolverHandle_t cusolverH = NULL;
	hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
	cusolver_status = hipsolverDnCreate(&cusolverH);
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

	// step 2: copy A and B to device
	const int lda = rows;
	hipError_t cudaStat = hipSuccess;
	float * d_A = NULL;
	cudaStat = hipMalloc((void**)&d_A, sizeof(float) * lda * cols);
	assert(hipSuccess == cudaStat);
	float *d_U = NULL;
	cudaStat = hipMalloc((void**)&d_U, sizeof(float) * lda * rows);
	assert(hipSuccess == cudaStat);
	float *d_S = NULL;
	cudaStat = hipMalloc((void**)&d_S, sizeof(float) * cols);
	assert(hipSuccess == cudaStat);
	float *d_VT = NULL;
	cudaStat = hipMalloc((void**)&d_VT, sizeof(float) * lda * cols);
	assert(hipSuccess == cudaStat);
	int * devInfo = NULL;
	cudaStat = hipMalloc((void**)&devInfo, sizeof(int));
	assert(hipSuccess == cudaStat);

	cudaStat = hipMemcpy(d_A, matrix_A, sizeof(float) * lda * cols, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);

	row_major_2_col_major_device(lda, cols, d_A);

	// step 3: query working space of SVD
	int lwork = 0;
	cusolver_status = hipsolverDnDgesvd_bufferSize(
		cusolverH,
		rows,
		cols,
		&lwork);
	assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);

	float *d_work = NULL;
	cudaStat = hipMalloc((void**)&d_work, sizeof(float) * lwork);
	assert(hipSuccess == cudaStat);

	// step 4: compute SVD
	signed char jobu = 'A'; // all m columns of U
	signed char jobvt = 'A'; // all n columns of VT
	float *d_rwork = NULL;
	cusolver_status = hipsolverDnSgesvd(
		cusolverH,
		jobu,
		jobvt,
		rows,
		cols,
		d_A,
		lda,
		d_S,
		d_U,
		lda,  // ldu
		d_VT,
		lda, // ldvt,
		d_work,
		lwork,
		d_rwork,
		devInfo);
	cudaStat = hipDeviceSynchronize();
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
	assert(hipSuccess == cudaStat);

	// step 5: copy back
	if (matrix_U) {
		col_major_2_row_major_device(lda, k, d_U);
		cudaStat = hipMemcpy(matrix_U, d_U, sizeof(float) * lda * k, hipMemcpyDeviceToHost);
		assert(hipSuccess == cudaStat);
	}
	if (array_S) {
		cudaStat = hipMemcpy(array_S, d_S, sizeof(float) * k, hipMemcpyDeviceToHost);
		assert(hipSuccess == cudaStat);
	}
	if (matrix_VT) {
		float * d_Temp = NULL;
		cudaStat = hipMalloc((void**)&d_Temp, sizeof(float) * k * cols);
		assert(hipSuccess == cudaStat);
		for (int i = 0; i < cols; i++) {
			cudaStat = hipMemcpy(d_Temp + i * k, d_VT + i * lda, sizeof(float) * k, hipMemcpyDeviceToDevice);
			assert(hipSuccess == cudaStat);
		}
		col_major_2_row_major_device(k, cols, d_Temp);
		cudaStat = hipMemcpy(matrix_VT, d_Temp, sizeof(float) * k * cols, hipMemcpyDeviceToHost);
		assert(hipSuccess == cudaStat);
		if (d_Temp) hipFree(d_Temp);
	}
	int info_gpu = 0;
	cudaStat = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
	assert(hipSuccess == cudaStat);
	assert(0 == info_gpu);

	// step 6: free
	if (d_A) hipFree(d_A);
	if (d_S) hipFree(d_S);
	if (d_U) hipFree(d_U);
	if (d_VT) hipFree(d_VT);
	if (devInfo) hipFree(devInfo);
	if (d_work) hipFree(d_work);
	if (d_rwork) hipFree(d_rwork);
	if (cusolverH) hipsolverDnDestroy(cusolverH);

	hipDeviceReset();
}

void multiplyByDiagonalMatrix(int rows, int cols, float * matrix, float * array) {
	// copy
	hipError_t cudaStat = hipSuccess;
	float *d_Matrix = NULL;
	cudaStat = hipMalloc((void**)&d_Matrix, sizeof(float) * rows * cols);
	assert(hipSuccess == cudaStat);
	float *d_Array = NULL;
	cudaStat = hipMalloc((void**)&d_Array, sizeof(float) * cols);
	assert(hipSuccess == cudaStat);
	float * d_Result = NULL;  // W = S*VT
	cudaStat = hipMalloc((void**)&d_Result, sizeof(float) * rows * cols);
	assert(hipSuccess == cudaStat);

	cudaStat = hipMemcpy(d_Matrix, matrix, sizeof(float) * rows * cols, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);
	cudaStat = hipMemcpy(d_Array, array, sizeof(float) * cols, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);

	row_major_2_col_major_device(rows, cols, d_Matrix);

	// create cublas handle
	hipblasHandle_t cublasH = NULL;
	hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
	cublas_status = hipblasCreate(&cublasH);
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	// R = M*S
	cublas_status = hipblasSdgmm(
		cublasH,
		HIPBLAS_SIDE_RIGHT,
		rows,
		cols,
		d_Matrix,
		rows,
		d_Array,
		1,
		d_Result,
		rows);
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	col_major_2_row_major_device(rows, cols, d_Result);

	// copy
	cudaStat = hipMemcpy(matrix, d_Result, sizeof(float) * rows * cols, hipMemcpyDeviceToHost);
	assert(hipSuccess == cudaStat);

	// free
	if (d_Array) hipFree(d_Array);
	if (d_Matrix) hipFree(d_Matrix);
	if (cublasH) hipblasDestroy(cublasH);

	hipDeviceReset();
}

void multiply(int rows, int k, int cols, float * matrixA, float * matrixB, float * result) {
	// copy
	hipError_t cudaStat = hipSuccess;
	float *d_A = NULL;
	cudaStat = hipMalloc((void**)&d_A, sizeof(float) * rows * k);
	assert(hipSuccess == cudaStat);
	float * d_B = NULL;
	cudaStat = hipMalloc((void**)&d_B, sizeof(float) * k * cols);
	assert(hipSuccess == cudaStat);
	float * d_Result = NULL;
	cudaStat = hipMalloc((void**)&d_Result, sizeof(float) * rows * cols);
	assert(hipSuccess == cudaStat);

	cudaStat = hipMemcpy(d_A, matrixA, sizeof(float) * rows * k, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);
	cudaStat = hipMemcpy(d_B, matrixB, sizeof(float) * k * cols, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);

	row_major_2_col_major_device(rows, k, d_A);
	row_major_2_col_major_device(k, cols, d_B);

	// create cublas handle
	hipblasHandle_t cublasH = NULL;
	hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
	cublas_status = hipblasCreate(&cublasH);
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	// A := U*W
	assert(hipSuccess == cudaStat);
	const float h_one = 1;
	const float h_zero = 0;
	cublas_status = hipblasSgemm(
		cublasH,
		HIPBLAS_OP_N,
		HIPBLAS_OP_N,
		rows,
		cols,
		k,
		&h_one,
		d_A,
		rows,
		d_B,
		k,
		&h_zero,
		d_Result,
		rows);
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	// copy
	col_major_2_row_major_device(rows, cols, d_Result);

	cudaStat = hipMemcpy(result, d_Result, sizeof(float) * rows * cols, hipMemcpyDeviceToHost);
	assert(hipSuccess == cudaStat);

	// free
	if (d_A) hipFree(d_A);
	if (d_B) hipFree(d_B);
	if (d_Result) hipFree(d_Result);
	if (cublasH) hipblasDestroy(cublasH);

	hipDeviceReset();
}