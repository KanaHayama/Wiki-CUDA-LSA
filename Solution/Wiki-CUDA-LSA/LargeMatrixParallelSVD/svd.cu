#include"svd.cuh"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>

void svd(int rows, int cols, double * matrix_A, double * matrix_U, double * array_S, double * matrix_VT) {
	
	// step 1: create cusolverDn handle
	hipsolverHandle_t cusolverH = NULL;
	hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
	cusolver_status = hipsolverDnCreate(&cusolverH);
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

	// step 2: copy A and B to device
	const int lda = rows;
	hipError_t cudaStat = hipSuccess;
	double * d_A = NULL;
	cudaStat = hipMalloc((void**)&d_A, sizeof(double) * lda * cols);
	assert(hipSuccess == cudaStat);
	double *d_U = NULL;
	cudaStat = hipMalloc((void**)&d_U, sizeof(double) * lda * rows);
	assert(hipSuccess == cudaStat);
	double *d_S = NULL;
	cudaStat = hipMalloc((void**)&d_S, sizeof(double) * cols);
	assert(hipSuccess == cudaStat);
	double *d_VT = NULL;
	cudaStat = hipMalloc((void**)&d_VT, sizeof(double) * lda * cols);
	assert(hipSuccess == cudaStat);
	int * devInfo = NULL;
	cudaStat = hipMalloc((void**)&devInfo, sizeof(int));
	assert(hipSuccess == cudaStat);

	cudaStat = hipMemcpy(d_A, matrix_A, sizeof(double) * lda * cols, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);

	// step 3: query working space of SVD
	int lwork = 0;
	cusolver_status = hipsolverDnDgesvd_bufferSize(
		cusolverH,
		rows,
		cols,
		&lwork);
	assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);

	double *d_work = NULL;
	cudaStat = hipMalloc((void**)&d_work, sizeof(double) * lwork);
	assert(hipSuccess == cudaStat);

	// step 4: compute SVD
	signed char jobu = 'A'; // all m columns of U
	signed char jobvt = 'A'; // all n columns of VT
	double *d_rwork = NULL;
	cusolver_status = hipsolverDnDgesvd(
		cusolverH,
		jobu,
		jobvt,
		rows,
		cols,
		d_A,
		lda,
		d_S,
		d_U,
		lda,  // ldu
		d_VT,
		lda, // ldvt,
		d_work,
		lwork,
		d_rwork,
		devInfo);
	cudaStat = hipDeviceSynchronize();
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
	assert(hipSuccess == cudaStat);

	// step 5: copy back
	if (matrix_U) {
		cudaStat = hipMemcpy(matrix_U, d_U, sizeof(double) * lda * rows, hipMemcpyDeviceToHost);
		assert(hipSuccess == cudaStat);
	}
	if (array_S) {
		cudaStat = hipMemcpy(array_S, d_S, sizeof(double) * cols, hipMemcpyDeviceToHost);
		assert(hipSuccess == cudaStat);
	}
	if (matrix_VT) {
		cudaStat = hipMemcpy(matrix_VT, d_VT, sizeof(double) * lda * cols, hipMemcpyDeviceToHost);
		assert(hipSuccess == cudaStat);
	}
	int info_gpu = 0;
	cudaStat = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
	assert(hipSuccess == cudaStat);
	assert(0 == info_gpu);

	// step 6: free
	if (d_A) hipFree(d_A);
	if (d_S) hipFree(d_S);
	if (d_U) hipFree(d_U);
	if (d_VT) hipFree(d_VT);
	if (devInfo) hipFree(devInfo);
	if (d_work) hipFree(d_work);
	if (d_rwork) hipFree(d_rwork);
	if (cusolverH) hipsolverDnDestroy(cusolverH);

	hipDeviceReset();
}

void svd_r(int rows, int cols, double * matrix_U, double * array_S, double * matrix_VT, double * matrix_A) {
	// copy
	const int lda = rows;
	hipError_t cudaStat = hipSuccess;
	double *d_U = NULL;
	cudaStat = hipMalloc((void**)&d_U, sizeof(double) * lda * rows);
	assert(hipSuccess == cudaStat);
	double *d_S = NULL;
	cudaStat = hipMalloc((void**)&d_S, sizeof(double) * cols);
	assert(hipSuccess == cudaStat);
	double *d_VT = NULL;
	cudaStat = hipMalloc((void**)&d_VT, sizeof(double) * lda * cols);
	assert(hipSuccess == cudaStat);
	double * d_W = NULL;  // W = S*VT
	cudaStat = hipMalloc((void**)&d_W, sizeof(double) * lda * cols);
	assert(hipSuccess == cudaStat);
	double * d_A = NULL;
	cudaStat = hipMalloc((void**)&d_A, sizeof(double) * lda * cols);
	assert(hipSuccess == cudaStat);
	int * devInfo = NULL;
	cudaStat = hipMalloc((void**)&devInfo, sizeof(int));
	assert(hipSuccess == cudaStat);

	cudaStat = hipMemcpy(d_U, matrix_U, sizeof(double) * lda * rows, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);
	cudaStat = hipMemcpy(d_S, array_S, sizeof(double) * cols, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);
	cudaStat = hipMemcpy(d_VT, matrix_VT, sizeof(double) * lda * cols, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);

	// create cublas handle
	hipblasHandle_t cublasH = NULL;
	hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
	cublas_status = hipblasCreate(&cublasH);
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	// W = S*VT
	cublas_status = hipblasDdgmm(
		cublasH,
		HIPBLAS_SIDE_LEFT,
		cols,
		cols,
		d_VT,
		lda,
		d_S,
		1,
		d_W,
		lda);
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	// A := U*W
	assert(hipSuccess == cudaStat);
	const double h_one = 1;
	const double h_zero = 0;
	cublas_status = hipblasDgemm(
		cublasH,
		HIPBLAS_OP_N, // U
		HIPBLAS_OP_N, // W
		rows, // number of rows of A
		cols, // number of columns of A
		cols, // number of columns of U 
		&h_one, /* host pointer */
		d_U, // U
		lda,
		d_W, // W
		lda,
		&h_zero, /* hostpointer */
		d_A,
		lda);
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	// copy
	if (matrix_A) {
		cudaStat = hipMemcpy(matrix_A, d_A, sizeof(double) * lda * cols, hipMemcpyDeviceToHost);
		assert(hipSuccess == cudaStat);
	}

	// free
	if (d_A) hipFree(d_A);
	if (d_S) hipFree(d_S);
	if (d_U) hipFree(d_U);
	if (d_VT) hipFree(d_VT);
	if (devInfo) hipFree(devInfo);
	if (cublasH) hipblasDestroy(cublasH);

	hipDeviceReset();
}

//TODO: use k to speed up svd, rather than apply after svd
void approximate_svd(int rows, int cols, int k, double * matrix_A, double * matrix_U, double * array_S, double * matrix_VT) {
	assert(k <= rows && k <= cols);
	// step 1: create cusolverDn handle
	hipsolverHandle_t cusolverH = NULL;
	hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
	cusolver_status = hipsolverDnCreate(&cusolverH);
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

	// step 2: copy A and B to device
	const int lda = rows;
	hipError_t cudaStat = hipSuccess;
	double * d_A = NULL;
	cudaStat = hipMalloc((void**)&d_A, sizeof(double) * lda * cols);
	assert(hipSuccess == cudaStat);
	double *d_U = NULL;
	cudaStat = hipMalloc((void**)&d_U, sizeof(double) * lda * rows);
	assert(hipSuccess == cudaStat);
	double *d_S = NULL;
	cudaStat = hipMalloc((void**)&d_S, sizeof(double) * cols);
	assert(hipSuccess == cudaStat);
	double *d_VT = NULL;
	cudaStat = hipMalloc((void**)&d_VT, sizeof(double) * lda * cols);
	assert(hipSuccess == cudaStat);
	int * devInfo = NULL;
	cudaStat = hipMalloc((void**)&devInfo, sizeof(int));
	assert(hipSuccess == cudaStat);

	cudaStat = hipMemcpy(d_A, matrix_A, sizeof(double) * lda * cols, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);

	// step 3: query working space of SVD
	int lwork = 0;
	cusolver_status = hipsolverDnDgesvd_bufferSize(
		cusolverH,
		rows,
		cols,
		&lwork);
	assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);

	double *d_work = NULL;
	cudaStat = hipMalloc((void**)&d_work, sizeof(double) * lwork);
	assert(hipSuccess == cudaStat);

	// step 4: compute SVD
	signed char jobu = 'A'; // all m columns of U
	signed char jobvt = 'A'; // all n columns of VT
	double *d_rwork = NULL;
	cusolver_status = hipsolverDnDgesvd(
		cusolverH,
		jobu,
		jobvt,
		rows,
		cols,
		d_A,
		lda,
		d_S,
		d_U,
		lda,  // ldu
		d_VT,
		lda, // ldvt,
		d_work,
		lwork,
		d_rwork,
		devInfo);
	cudaStat = hipDeviceSynchronize();
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
	assert(hipSuccess == cudaStat);

	// step 5: copy back
	if (matrix_U) {
		for (int i = 0; i < lda; i++) {
			cudaStat = hipMemcpy(matrix_U + i * k, d_U + i * rows, sizeof(double) * k, hipMemcpyDeviceToHost);
			assert(hipSuccess == cudaStat);
		}
	}
	if (array_S) {
		cudaStat = hipMemcpy(array_S, d_S, sizeof(double) * k, hipMemcpyDeviceToHost);
		assert(hipSuccess == cudaStat);
	}
	if (matrix_VT) {
		cudaStat = hipMemcpy(matrix_VT, d_VT, sizeof(double) * k * cols, hipMemcpyDeviceToHost);
		assert(hipSuccess == cudaStat);
	}
	int info_gpu = 0;
	cudaStat = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
	assert(hipSuccess == cudaStat);
	assert(0 == info_gpu);

	// step 6: free
	if (d_A) hipFree(d_A);
	if (d_S) hipFree(d_S);
	if (d_U) hipFree(d_U);
	if (d_VT) hipFree(d_VT);
	if (devInfo) hipFree(devInfo);
	if (d_work) hipFree(d_work);
	if (d_rwork) hipFree(d_rwork);
	if (cusolverH) hipsolverDnDestroy(cusolverH);

	hipDeviceReset();
}