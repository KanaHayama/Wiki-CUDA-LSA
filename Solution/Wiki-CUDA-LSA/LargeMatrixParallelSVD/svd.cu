#include"svd.cuh"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>

void transpose_device(int rows, int cols, double * d_Matrix) {
	hipError_t cudaStat = hipSuccess;
	double * d_Result = NULL;
	cudaStat = hipMalloc((void**)&d_Result, sizeof(double) * rows * cols);
	assert(hipSuccess == cudaStat);
	const double alpha = 1;
	const double beta = 0;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
	hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, rows, cols, &alpha, d_Matrix, cols, &beta, d_Matrix, rows, d_Result, rows);//hipblasSgeam: float version
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
	cudaStat = hipMemcpy(d_Matrix, d_Result, sizeof(double) * rows * cols, hipMemcpyDeviceToDevice);
	assert(hipSuccess == cudaStat);
	if (d_Result) hipFree(d_Result);
	hipblasDestroy(handle);
}

void transpose(int rows, int cols, double * matrix) {
	hipError_t cudaStat = hipSuccess;
	double * d_Matrix = NULL;
	cudaStat = hipMalloc((void**)&d_Matrix, sizeof(double) * rows * cols);
	assert(hipSuccess == cudaStat);
	cudaStat = hipMemcpy(d_Matrix, matrix, sizeof(double) * rows * cols, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);
	transpose_device(rows, cols, d_Matrix);
	cudaStat = hipMemcpy(matrix, d_Matrix, sizeof(double) * rows * cols, hipMemcpyDeviceToHost);
	assert(hipSuccess == cudaStat);
	if (d_Matrix) hipFree(d_Matrix);
}

inline void row_major_2_col_major_device(int rows, int cols, double * matrix) {
	transpose_device(rows, cols, matrix);
}

inline void col_major_2_row_major_device(int rows, int cols, double * matrix) {
	transpose_device(cols, rows, matrix);
}

void svd(int rows, int cols, double * matrix_A, double * matrix_U, double * array_S, double * matrix_VT) {
	assert(rows >= cols);
	// step 1: create cusolverDn handle
	hipsolverHandle_t cusolverH = NULL;
	hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
	cusolver_status = hipsolverDnCreate(&cusolverH);
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

	// step 2: copy A and B to device
	const int lda = rows;
	hipError_t cudaStat = hipSuccess;
	double * d_A = NULL;
	cudaStat = hipMalloc((void**)&d_A, sizeof(double) * lda * cols);
	assert(hipSuccess == cudaStat);
	double *d_U = NULL;
	cudaStat = hipMalloc((void**)&d_U, sizeof(double) * lda * rows);
	assert(hipSuccess == cudaStat);
	double *d_S = NULL;
	cudaStat = hipMalloc((void**)&d_S, sizeof(double) * cols);
	assert(hipSuccess == cudaStat);
	double *d_VT = NULL;
	cudaStat = hipMalloc((void**)&d_VT, sizeof(double) * lda * cols);
	assert(hipSuccess == cudaStat);
	int * devInfo = NULL;
	cudaStat = hipMalloc((void**)&devInfo, sizeof(int));
	assert(hipSuccess == cudaStat);

	cudaStat = hipMemcpy(d_A, matrix_A, sizeof(double) * lda * cols, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);

	row_major_2_col_major_device(lda, cols, d_A);

	// step 3: query working space of SVD
	int lwork = 0;
	cusolver_status = hipsolverDnDgesvd_bufferSize(
		cusolverH,
		rows,
		cols,
		&lwork);
	assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);

	double *d_work = NULL;
	cudaStat = hipMalloc((void**)&d_work, sizeof(double) * lwork);
	assert(hipSuccess == cudaStat);

	// step 4: compute SVD
	signed char jobu = 'A'; // all m columns of U
	signed char jobvt = 'A'; // all n columns of VT
	double *d_rwork = NULL;
	cusolver_status = hipsolverDnDgesvd(
		cusolverH,
		jobu,
		jobvt,
		rows,
		cols,
		d_A,
		lda,
		d_S,
		d_U,
		lda,  // ldu
		d_VT,
		lda, // ldvt,
		d_work,
		lwork,
		d_rwork,
		devInfo);
	cudaStat = hipDeviceSynchronize();
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
	assert(hipSuccess == cudaStat);

	// step 5: copy back
	if (matrix_U) {
		col_major_2_row_major_device(lda, rows, d_U);
		cudaStat = hipMemcpy(matrix_U, d_U, sizeof(double) * lda * rows, hipMemcpyDeviceToHost);
		assert(hipSuccess == cudaStat);
	}
	if (array_S) {
		cudaStat = hipMemcpy(array_S, d_S, sizeof(double) * cols, hipMemcpyDeviceToHost);
		assert(hipSuccess == cudaStat);
	}
	if (matrix_VT) {
		col_major_2_row_major_device(lda, cols, d_VT);
		cudaStat = hipMemcpy(matrix_VT, d_VT, sizeof(double) * lda * cols, hipMemcpyDeviceToHost);
		assert(hipSuccess == cudaStat);
	}
	int info_gpu = 0;
	cudaStat = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
	assert(hipSuccess == cudaStat);
	assert(0 == info_gpu);

	// step 6: free
	if (d_A) hipFree(d_A);
	if (d_S) hipFree(d_S);
	if (d_U) hipFree(d_U);
	if (d_VT) hipFree(d_VT);
	if (devInfo) hipFree(devInfo);
	if (d_work) hipFree(d_work);
	if (d_rwork) hipFree(d_rwork);
	if (cusolverH) hipsolverDnDestroy(cusolverH);

	hipDeviceReset();
}

void svd_r(int rows, int cols, double * matrix_U, double * array_S, double * matrix_VT, double * matrix_A) {
	// copy
	const int lda = rows;
	hipError_t cudaStat = hipSuccess;
	double *d_U = NULL;
	cudaStat = hipMalloc((void**)&d_U, sizeof(double) * lda * rows);
	assert(hipSuccess == cudaStat);
	double *d_S = NULL;
	cudaStat = hipMalloc((void**)&d_S, sizeof(double) * cols);
	assert(hipSuccess == cudaStat);
	double *d_VT = NULL;
	cudaStat = hipMalloc((void**)&d_VT, sizeof(double) * lda * cols);
	assert(hipSuccess == cudaStat);
	double * d_W = NULL;  // W = S*VT
	cudaStat = hipMalloc((void**)&d_W, sizeof(double) * lda * cols);
	assert(hipSuccess == cudaStat);
	double * d_A = NULL;
	cudaStat = hipMalloc((void**)&d_A, sizeof(double) * lda * cols);
	assert(hipSuccess == cudaStat);
	int * devInfo = NULL;
	cudaStat = hipMalloc((void**)&devInfo, sizeof(int));
	assert(hipSuccess == cudaStat);

	cudaStat = hipMemcpy(d_U, matrix_U, sizeof(double) * lda * rows, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);
	cudaStat = hipMemcpy(d_S, array_S, sizeof(double) * cols, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);
	cudaStat = hipMemcpy(d_VT, matrix_VT, sizeof(double) * lda * cols, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);

	// create cublas handle
	hipblasHandle_t cublasH = NULL;
	hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
	cublas_status = hipblasCreate(&cublasH);
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	// W = S*VT
	cublas_status = hipblasDdgmm(
		cublasH,
		HIPBLAS_SIDE_LEFT,
		cols,
		cols,
		d_VT,
		lda,
		d_S,
		1,
		d_W,
		lda);
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	// A := U*W
	assert(hipSuccess == cudaStat);
	const double h_one = 1;
	const double h_zero = 0;
	cublas_status = hipblasDgemm(
		cublasH,
		HIPBLAS_OP_N, // U
		HIPBLAS_OP_N, // W
		rows, // number of rows of A
		cols, // number of columns of A
		cols, // number of columns of U 
		&h_one, /* host pointer */
		d_U, // U
		lda,
		d_W, // W
		lda,
		&h_zero, /* hostpointer */
		d_A,
		lda);
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	// copy
	if (matrix_A) {
		cudaStat = hipMemcpy(matrix_A, d_A, sizeof(double) * lda * cols, hipMemcpyDeviceToHost);
		assert(hipSuccess == cudaStat);
	}

	// free
	if (d_A) hipFree(d_A);
	if (d_S) hipFree(d_S);
	if (d_U) hipFree(d_U);
	if (d_VT) hipFree(d_VT);
	if (devInfo) hipFree(devInfo);
	if (cublasH) hipblasDestroy(cublasH);

	hipDeviceReset();
}

//TODO: use k to speed up svd, rather than apply after svd
void approximate_svd(int rows, int cols, int k, double * matrix_A, double * matrix_U, double * array_S, double * matrix_VT) {
	assert(rows >= cols);
	assert(k <= rows && k <= cols);
	// step 1: create cusolverDn handle
	hipsolverHandle_t cusolverH = NULL;
	hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
	cusolver_status = hipsolverDnCreate(&cusolverH);
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

	// step 2: copy A and B to device
	const int lda = rows;
	hipError_t cudaStat = hipSuccess;
	double * d_A = NULL;
	cudaStat = hipMalloc((void**)&d_A, sizeof(double) * lda * cols);
	assert(hipSuccess == cudaStat);
	double *d_U = NULL;
	cudaStat = hipMalloc((void**)&d_U, sizeof(double) * lda * rows);
	assert(hipSuccess == cudaStat);
	double *d_S = NULL;
	cudaStat = hipMalloc((void**)&d_S, sizeof(double) * cols);
	assert(hipSuccess == cudaStat);
	double *d_VT = NULL;
	cudaStat = hipMalloc((void**)&d_VT, sizeof(double) * lda * cols);
	assert(hipSuccess == cudaStat);
	int * devInfo = NULL;
	cudaStat = hipMalloc((void**)&devInfo, sizeof(int));
	assert(hipSuccess == cudaStat);

	cudaStat = hipMemcpy(d_A, matrix_A, sizeof(double) * lda * cols, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);

	row_major_2_col_major_device(lda, cols, d_A);

	// step 3: query working space of SVD
	int lwork = 0;
	cusolver_status = hipsolverDnDgesvd_bufferSize(
		cusolverH,
		rows,
		cols,
		&lwork);
	assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);

	double *d_work = NULL;
	cudaStat = hipMalloc((void**)&d_work, sizeof(double) * lwork);
	assert(hipSuccess == cudaStat);

	// step 4: compute SVD
	signed char jobu = 'A'; // all m columns of U
	signed char jobvt = 'A'; // all n columns of VT
	double *d_rwork = NULL;
	cusolver_status = hipsolverDnDgesvd(
		cusolverH,
		jobu,
		jobvt,
		rows,
		cols,
		d_A,
		lda,
		d_S,
		d_U,
		lda,  // ldu
		d_VT,
		lda, // ldvt,
		d_work,
		lwork,
		d_rwork,
		devInfo);
	cudaStat = hipDeviceSynchronize();
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
	assert(hipSuccess == cudaStat);

	// step 5: copy back
	if (matrix_U) {
		col_major_2_row_major_device(lda, k, d_U);
		cudaStat = hipMemcpy(matrix_U, d_U, sizeof(double) * lda * k, hipMemcpyDeviceToHost);
		assert(hipSuccess == cudaStat);
	}
	if (array_S) {
		cudaStat = hipMemcpy(array_S, d_S, sizeof(double) * k, hipMemcpyDeviceToHost);
		assert(hipSuccess == cudaStat);
	}
	if (matrix_VT) {
		double * d_Temp = NULL;
		cudaStat = hipMalloc((void**)&d_Temp, sizeof(double) * k * cols);
		assert(hipSuccess == cudaStat);
		for (int i = 0; i < cols; i++) {
			cudaStat = hipMemcpy(d_Temp + i * k, d_VT + i * lda, sizeof(double) * k, hipMemcpyDeviceToDevice);
			assert(hipSuccess == cudaStat);
		}
		col_major_2_row_major_device(k, cols, d_Temp);
		cudaStat = hipMemcpy(matrix_VT, d_Temp, sizeof(double) * k * cols, hipMemcpyDeviceToHost);
		assert(hipSuccess == cudaStat);
		if (d_Temp) hipFree(d_Temp);
	}
	int info_gpu = 0;
	cudaStat = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
	assert(hipSuccess == cudaStat);
	assert(0 == info_gpu);

	// step 6: free
	if (d_A) hipFree(d_A);
	if (d_S) hipFree(d_S);
	if (d_U) hipFree(d_U);
	if (d_VT) hipFree(d_VT);
	if (devInfo) hipFree(devInfo);
	if (d_work) hipFree(d_work);
	if (d_rwork) hipFree(d_rwork);
	if (cusolverH) hipsolverDnDestroy(cusolverH);

	hipDeviceReset();
}

void multiplyByDiagonalMatrix(int rows, int cols, double * matrix, double * array) {
	// copy
	hipError_t cudaStat = hipSuccess;
	double *d_Matrix = NULL;
	cudaStat = hipMalloc((void**)&d_Matrix, sizeof(double) * rows * cols);
	assert(hipSuccess == cudaStat);
	double *d_Array = NULL;
	cudaStat = hipMalloc((void**)&d_Array, sizeof(double) * cols);
	assert(hipSuccess == cudaStat);
	double * d_Result = NULL;  // W = S*VT
	cudaStat = hipMalloc((void**)&d_Result, sizeof(double) * rows * cols);
	assert(hipSuccess == cudaStat);

	cudaStat = hipMemcpy(d_Matrix, matrix, sizeof(double) * rows * cols, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);
	cudaStat = hipMemcpy(d_Array, array, sizeof(double) * cols, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);

	row_major_2_col_major_device(rows, cols, d_Matrix);

	// create cublas handle
	hipblasHandle_t cublasH = NULL;
	hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
	cublas_status = hipblasCreate(&cublasH);
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	// R = M*S
	cublas_status = hipblasDdgmm(
		cublasH,
		HIPBLAS_SIDE_RIGHT,
		rows,
		cols,
		d_Matrix,
		rows,
		d_Array,
		1,
		d_Result,
		rows);
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	col_major_2_row_major_device(rows, cols, d_Result);

	// copy
	cudaStat = hipMemcpy(matrix, d_Result, sizeof(double) * rows * cols, hipMemcpyDeviceToHost);
	assert(hipSuccess == cudaStat);

	// free
	if (d_Array) hipFree(d_Array);
	if (d_Matrix) hipFree(d_Matrix);
	if (cublasH) hipblasDestroy(cublasH);

	hipDeviceReset();
}

void multiply(int rows, int k, int cols, double * matrixA, double * matrixB, double * result) {
	// copy
	hipError_t cudaStat = hipSuccess;
	double *d_A = NULL;
	cudaStat = hipMalloc((void**)&d_A, sizeof(double) * rows * k);
	assert(hipSuccess == cudaStat);
	double * d_B = NULL;
	cudaStat = hipMalloc((void**)&d_B, sizeof(double) * k * cols);
	assert(hipSuccess == cudaStat);
	double * d_Result = NULL;
	cudaStat = hipMalloc((void**)&d_Result, sizeof(double) * rows * cols);
	assert(hipSuccess == cudaStat);

	cudaStat = hipMemcpy(d_A, matrixA, sizeof(double) * rows * k, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);
	cudaStat = hipMemcpy(d_B, matrixB, sizeof(double) * k * cols, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);

	row_major_2_col_major_device(rows, k, d_A);
	row_major_2_col_major_device(k, cols, d_B);

	// create cublas handle
	hipblasHandle_t cublasH = NULL;
	hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
	cublas_status = hipblasCreate(&cublasH);
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	// A := U*W
	assert(hipSuccess == cudaStat);
	const double h_one = 1;
	const double h_zero = 0;
	cublas_status = hipblasDgemm(
		cublasH,
		HIPBLAS_OP_N,
		HIPBLAS_OP_N,
		rows,
		k,
		cols,
		&h_one,
		d_A,
		rows,
		d_B,
		k,
		&h_zero,
		d_Result,
		rows);
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	// copy
	col_major_2_row_major_device(rows, cols, d_Result);

	cudaStat = hipMemcpy(result, d_Result, sizeof(double) * rows * cols, hipMemcpyDeviceToHost);
	assert(hipSuccess == cudaStat);

	// free
	if (d_A) hipFree(d_A);
	if (d_B) hipFree(d_B);
	if (d_Result) hipFree(d_Result);
	if (cublasH) hipblasDestroy(cublasH);

	hipDeviceReset();
}