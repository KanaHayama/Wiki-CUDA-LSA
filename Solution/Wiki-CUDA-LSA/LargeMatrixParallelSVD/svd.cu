#include"svd.cuh"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>

//TODO: parallelize
void row_major_2_col_major(int rows, int cols, double * matrix) {
	double * temp = new double[rows * cols];
	memcpy(temp, matrix, sizeof(double) * rows * cols);
	for (int row = 0; row < rows; row++) {
		for (int col = 0; col < cols; col++) {
			matrix[col * rows + row] = temp[row * cols + col];
		}
	}
	delete[] temp;
}

//TODO: parallelize
void col_major_2_row_major(int rows, int cols, double * matrix) {
	double * temp = new double[rows * cols];
	memcpy(temp, matrix, sizeof(double) * rows * cols);
	for (int row = 0; row < rows; row++) {
		for (int col = 0; col < cols; col++) {
			matrix[row * cols + col] = temp[col * rows + row];
		}
	}
	delete[] temp;
}

void svd(int rows, int cols, double * matrix_A, double * matrix_U, double * array_S, double * matrix_VT) {
	assert(rows >= cols);
	row_major_2_col_major(rows, cols, matrix_A);
	// step 1: create cusolverDn handle
	hipsolverHandle_t cusolverH = NULL;
	hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
	cusolver_status = hipsolverDnCreate(&cusolverH);
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

	// step 2: copy A and B to device
	const int lda = rows;
	hipError_t cudaStat = hipSuccess;
	double * d_A = NULL;
	cudaStat = hipMalloc((void**)&d_A, sizeof(double) * lda * cols);
	assert(hipSuccess == cudaStat);
	double *d_U = NULL;
	cudaStat = hipMalloc((void**)&d_U, sizeof(double) * lda * rows);
	assert(hipSuccess == cudaStat);
	double *d_S = NULL;
	cudaStat = hipMalloc((void**)&d_S, sizeof(double) * cols);
	assert(hipSuccess == cudaStat);
	double *d_VT = NULL;
	cudaStat = hipMalloc((void**)&d_VT, sizeof(double) * lda * cols);
	assert(hipSuccess == cudaStat);
	int * devInfo = NULL;
	cudaStat = hipMalloc((void**)&devInfo, sizeof(int));
	assert(hipSuccess == cudaStat);

	cudaStat = hipMemcpy(d_A, matrix_A, sizeof(double) * lda * cols, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);

	// step 3: query working space of SVD
	int lwork = 0;
	cusolver_status = hipsolverDnDgesvd_bufferSize(
		cusolverH,
		rows,
		cols,
		&lwork);
	assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);

	double *d_work = NULL;
	cudaStat = hipMalloc((void**)&d_work, sizeof(double) * lwork);
	assert(hipSuccess == cudaStat);

	// step 4: compute SVD
	signed char jobu = 'A'; // all m columns of U
	signed char jobvt = 'A'; // all n columns of VT
	double *d_rwork = NULL;
	cusolver_status = hipsolverDnDgesvd(
		cusolverH,
		jobu,
		jobvt,
		rows,
		cols,
		d_A,
		lda,
		d_S,
		d_U,
		lda,  // ldu
		d_VT,
		lda, // ldvt,
		d_work,
		lwork,
		d_rwork,
		devInfo);
	cudaStat = hipDeviceSynchronize();
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
	assert(hipSuccess == cudaStat);

	// step 5: copy back
	if (matrix_U) {
		cudaStat = hipMemcpy(matrix_U, d_U, sizeof(double) * lda * rows, hipMemcpyDeviceToHost);
		assert(hipSuccess == cudaStat);
		col_major_2_row_major(lda, rows, matrix_U);
	}
	if (array_S) {
		cudaStat = hipMemcpy(array_S, d_S, sizeof(double) * cols, hipMemcpyDeviceToHost);
		assert(hipSuccess == cudaStat);
	}
	if (matrix_VT) {
		cudaStat = hipMemcpy(matrix_VT, d_VT, sizeof(double) * lda * cols, hipMemcpyDeviceToHost);
		assert(hipSuccess == cudaStat);
		col_major_2_row_major(lda, cols, matrix_VT);
	}
	int info_gpu = 0;
	cudaStat = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
	assert(hipSuccess == cudaStat);
	assert(0 == info_gpu);

	// step 6: free
	if (d_A) hipFree(d_A);
	if (d_S) hipFree(d_S);
	if (d_U) hipFree(d_U);
	if (d_VT) hipFree(d_VT);
	if (devInfo) hipFree(devInfo);
	if (d_work) hipFree(d_work);
	if (d_rwork) hipFree(d_rwork);
	if (cusolverH) hipsolverDnDestroy(cusolverH);

	hipDeviceReset();
}

void svd_r(int rows, int cols, double * matrix_U, double * array_S, double * matrix_VT, double * matrix_A) {
	// copy
	const int lda = rows;
	hipError_t cudaStat = hipSuccess;
	double *d_U = NULL;
	cudaStat = hipMalloc((void**)&d_U, sizeof(double) * lda * rows);
	assert(hipSuccess == cudaStat);
	double *d_S = NULL;
	cudaStat = hipMalloc((void**)&d_S, sizeof(double) * cols);
	assert(hipSuccess == cudaStat);
	double *d_VT = NULL;
	cudaStat = hipMalloc((void**)&d_VT, sizeof(double) * lda * cols);
	assert(hipSuccess == cudaStat);
	double * d_W = NULL;  // W = S*VT
	cudaStat = hipMalloc((void**)&d_W, sizeof(double) * lda * cols);
	assert(hipSuccess == cudaStat);
	double * d_A = NULL;
	cudaStat = hipMalloc((void**)&d_A, sizeof(double) * lda * cols);
	assert(hipSuccess == cudaStat);
	int * devInfo = NULL;
	cudaStat = hipMalloc((void**)&devInfo, sizeof(int));
	assert(hipSuccess == cudaStat);

	cudaStat = hipMemcpy(d_U, matrix_U, sizeof(double) * lda * rows, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);
	cudaStat = hipMemcpy(d_S, array_S, sizeof(double) * cols, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);
	cudaStat = hipMemcpy(d_VT, matrix_VT, sizeof(double) * lda * cols, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);

	// create cublas handle
	hipblasHandle_t cublasH = NULL;
	hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
	cublas_status = hipblasCreate(&cublasH);
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	// W = S*VT
	cublas_status = hipblasDdgmm(
		cublasH,
		HIPBLAS_SIDE_LEFT,
		cols,
		cols,
		d_VT,
		lda,
		d_S,
		1,
		d_W,
		lda);
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	// A := U*W
	assert(hipSuccess == cudaStat);
	const double h_one = 1;
	const double h_zero = 0;
	cublas_status = hipblasDgemm(
		cublasH,
		HIPBLAS_OP_N, // U
		HIPBLAS_OP_N, // W
		rows, // number of rows of A
		cols, // number of columns of A
		cols, // number of columns of U 
		&h_one, /* host pointer */
		d_U, // U
		lda,
		d_W, // W
		lda,
		&h_zero, /* hostpointer */
		d_A,
		lda);
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

	// copy
	if (matrix_A) {
		cudaStat = hipMemcpy(matrix_A, d_A, sizeof(double) * lda * cols, hipMemcpyDeviceToHost);
		assert(hipSuccess == cudaStat);
	}

	// free
	if (d_A) hipFree(d_A);
	if (d_S) hipFree(d_S);
	if (d_U) hipFree(d_U);
	if (d_VT) hipFree(d_VT);
	if (devInfo) hipFree(devInfo);
	if (cublasH) hipblasDestroy(cublasH);

	hipDeviceReset();
}

//TODO: use k to speed up svd, rather than apply after svd
void approximate_svd(int rows, int cols, int k, double * matrix_A, double * matrix_U, double * array_S, double * matrix_VT) {
	assert(rows >= cols);
	assert(k <= rows && k <= cols);
	row_major_2_col_major(rows, cols, matrix_A);
	// step 1: create cusolverDn handle
	hipsolverHandle_t cusolverH = NULL;
	hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
	cusolver_status = hipsolverDnCreate(&cusolverH);
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

	// step 2: copy A and B to device
	const int lda = rows;
	hipError_t cudaStat = hipSuccess;
	double * d_A = NULL;
	cudaStat = hipMalloc((void**)&d_A, sizeof(double) * lda * cols);
	assert(hipSuccess == cudaStat);
	double *d_U = NULL;
	cudaStat = hipMalloc((void**)&d_U, sizeof(double) * lda * rows);
	assert(hipSuccess == cudaStat);
	double *d_S = NULL;
	cudaStat = hipMalloc((void**)&d_S, sizeof(double) * cols);
	assert(hipSuccess == cudaStat);
	double *d_VT = NULL;
	cudaStat = hipMalloc((void**)&d_VT, sizeof(double) * lda * cols);
	assert(hipSuccess == cudaStat);
	int * devInfo = NULL;
	cudaStat = hipMalloc((void**)&devInfo, sizeof(int));
	assert(hipSuccess == cudaStat);

	cudaStat = hipMemcpy(d_A, matrix_A, sizeof(double) * lda * cols, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat);

	// step 3: query working space of SVD
	int lwork = 0;
	cusolver_status = hipsolverDnDgesvd_bufferSize(
		cusolverH,
		rows,
		cols,
		&lwork);
	assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);

	double *d_work = NULL;
	cudaStat = hipMalloc((void**)&d_work, sizeof(double) * lwork);
	assert(hipSuccess == cudaStat);

	// step 4: compute SVD
	signed char jobu = 'A'; // all m columns of U
	signed char jobvt = 'A'; // all n columns of VT
	double *d_rwork = NULL;
	cusolver_status = hipsolverDnDgesvd(
		cusolverH,
		jobu,
		jobvt,
		rows,
		cols,
		d_A,
		lda,
		d_S,
		d_U,
		lda,  // ldu
		d_VT,
		lda, // ldvt,
		d_work,
		lwork,
		d_rwork,
		devInfo);
	cudaStat = hipDeviceSynchronize();
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
	assert(hipSuccess == cudaStat);

	// step 5: copy back
	if (matrix_U) {
		cudaStat = hipMemcpy(matrix_U, d_U, sizeof(double) * lda * k, hipMemcpyDeviceToHost);
		assert(hipSuccess == cudaStat);
		col_major_2_row_major(lda, k, matrix_U);
	}
	if (array_S) {
		cudaStat = hipMemcpy(array_S, d_S, sizeof(double) * k, hipMemcpyDeviceToHost);
		assert(hipSuccess == cudaStat);
	}
	if (matrix_VT) {
		for (int i = 0; i < cols; i++) {
			cudaStat = hipMemcpy(matrix_VT + i * k, d_VT + i * lda, sizeof(double) * k, hipMemcpyDeviceToHost);
			assert(hipSuccess == cudaStat);
		}
		col_major_2_row_major(k, cols, matrix_VT);
	}
	int info_gpu = 0;
	cudaStat = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
	assert(hipSuccess == cudaStat);
	assert(0 == info_gpu);

	// step 6: free
	if (d_A) hipFree(d_A);
	if (d_S) hipFree(d_S);
	if (d_U) hipFree(d_U);
	if (d_VT) hipFree(d_VT);
	if (devInfo) hipFree(devInfo);
	if (d_work) hipFree(d_work);
	if (d_rwork) hipFree(d_rwork);
	if (cusolverH) hipsolverDnDestroy(cusolverH);

	hipDeviceReset();
}