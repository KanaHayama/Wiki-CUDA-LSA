#include "hip/hip_runtime.h"

#include<iostream>
#include<map>
#include<tfidf.cuh>
#include<svd.cuh>
#include"lsa.cuh"

const auto test_TD_docs = 12;
const auto test_TD_terms = 9;

static float test_doc_term_freq[test_TD_docs * test_TD_terms] = {
	1, 0, 0, 1, 0, 0, 0, 0, 0,
	1, 0, 1, 0, 0, 0, 0, 0, 0,
	1, 1, 0, 0, 0, 0, 0, 0, 0,
	0, 1, 1, 0, 1, 0, 0, 0, 0,
	0, 1, 1, 2, 0, 0, 0, 0, 0,
	0, 1, 0, 0, 1, 0, 0, 0, 0,
	0, 1, 0, 0, 1, 0, 0, 0, 0,
	0, 0, 1, 1, 0, 0, 0, 0, 0,
	0, 1, 0, 0, 0, 0, 0, 0, 1,
	0, 0, 0, 0, 0, 1, 1, 1, 0,
	0, 0, 0, 0, 0, 0, 1, 1, 1,
	0, 0, 0, 0, 0, 0, 0, 1, 1,
};

const auto test_k = 2;

const auto numConcept = 2;
const auto numTerms = 2;
const auto numDocs = 2;

void printColMajorMatrix(int m, int n, const float*A, int lda, const char* name) {
	for (int row = 0; row < m; row++) {
		for (int col = 0; col < n; col++) {
			float Areg = A[col * m + row];
			printf("%s(%d,%d) = %f\n", name, row, col, Areg);
		}
	}
}

void printRowMajorMatrix(int m, int n, const float*A, const char* name) {
	for (int row = 0; row < m; row++) {
		for (int col = 0; col < n; col++) {
			float Areg = A[row * n + col];
			printf("%s(%d,%d) = %f\n", name, row, col, Areg);
		}
	}
}

void printConcepts(const int numConcept, const int numTerms, const int numDocs, const std::vector<std::vector<std::tuple<int, float>>> topTerms, const std::vector<std::vector<std::tuple<int, float>>> topDocs) {
	for (int c = 0; c < numConcept; c++) {
		std::cout << "Concept " << c << ":" << std::endl;
		std::cout << "\t" << "Top Term Idx: ";
		for (int i = 0; i < numTerms; i++) {
			std::cout << std::get<0>(topTerms[c][i]) << "(" << std::get<1>(topTerms[c][i]) << "), ";
		}
		std::cout << std::endl;
		std::cout << "\t" << "Top Doc Idx: ";
		for (int i = 0; i < numDocs; i++) {
			std::cout << std::get<0>(topDocs[c][i]) << "(" << std::get<1>(topDocs[c][i]) << "), ";
		}
		std::cout << std::endl;
	}
}

void printTermTerm(const int lookUpIdx, const std::vector<std::tuple<int, float>> termTerm) {
	std::cout << "Term correlation to term index " << lookUpIdx << " ranking:" << std::endl;
	std::cout << "\t";
	for (auto & t : termTerm) {
		std::cout << std::get<0>(t) << "(" << std::get<1>(t) << "), ";
	}
	std::cout << std::endl;
}

void routine() {
	const int m = test_TD_docs;
	const int n = test_TD_terms;
	const int k = test_k;
	const int lda = m;
	//TF/IDF
	tfidf(m, n, test_doc_term_freq);
	//svd
	float U[lda * k];
	float S[k];
	float VT[k * n];
	printf("A = \n");
	printColMajorMatrix(m, n, test_doc_term_freq, lda, "A");
	printf("=====\n");
	approximate_svd(m, n, k, test_doc_term_freq, U, S, VT);
	printf("U = \n");
	printRowMajorMatrix(m, k, U, "U");
	printf("=====\n");
	printf("S = \n");
	printRowMajorMatrix(k, 1, S, "S");
	printf("=====\n");
	printf("VT = \n");
	printRowMajorMatrix(k, n, VT, "VT");
	printf("=====\n");
	printf("V = \n");
	float V[n * k];
	memcpy(V, VT, sizeof(float) * k * n);
	transpose(k, n, V);
	printRowMajorMatrix(n, k, V, "V");
	printf("=====\n");
	//concepts
	auto topTerms = topElementsInTopConcepts(n, k, V, numConcept, numTerms);
	auto topDocs = topElementsInTopConcepts(m, k, U, numConcept, numDocs);
	printConcepts(numConcept, numTerms, numDocs, topTerms, topDocs);
	printf("=====\n");
	//corrolated
	printf("V*S = \n");
	float VS[n * k];
	memcpy(VS, V, sizeof(float) * k * n);
	multiplyByDiagonalMatrix(n, k, VS, S);
	printRowMajorMatrix(n, k, VS, "VS");
	printf("=====\n");

	printf("norm(V*S) = \n");
	float normVS[n * k];
	memcpy(normVS, VS, sizeof(float) * k * n);
	rowsNormalized(n, k, normVS);
	printRowMajorMatrix(n, k, normVS, "normVS");
	printf("=====\n");

	printf("U*S = \n");
	float US[m * k];
	memcpy(US, U, sizeof(float) * k * m);
	multiplyByDiagonalMatrix(m, k, US, S);
	printRowMajorMatrix(m, k, US, "US");
	printf("=====\n");

	printf("norm(U*S) = \n");
	float normUS[m * k];
	memcpy(normUS, US, sizeof(float) * k * m);
	rowsNormalized(m, k, normUS);
	printRowMajorMatrix(m, k, normUS, "normUS");
	printf("=====\n");

	for (int lookUpTermIdx = 0; lookUpTermIdx < 3; lookUpTermIdx++) {
		auto termTerm = topsForTerm(n, k, normVS, n, k, normVS, lookUpTermIdx);
		printTermTerm(lookUpTermIdx, termTerm);
	}
}
/*
int main() {
	routine();
	return 0;
}
*/