#include "hip/hip_runtime.h"

#include<iostream>
#include<fstream>
#include<vector>
#include<tfidf.cuh>
#include<svd.cuh>
#include<string>
#include<cassert>
#include<algorithm>
#include<memory>
#include"lsa.cuh"

using std::vector;
using std::string;
using std::unique_ptr;
using std::shared_ptr;
using std::get;
using std::cout;
using std::endl;

//#define PRINT_MATRIX

void printMatrix(const char* name, int m, int n, const float*A) {
#ifdef PRINT_MATRIX
	std::cout << name << ":" << std::endl;
	for (int row = 0; row < m; row++) {
		for (int col = 0; col < n; col++) {
			float Areg = A[row * n + col];
			std::cout.width(15);
			std::cout << Areg;
		}
		std::cout << std::endl;
	}
#endif // PRINT_MATRIX
}

shared_ptr<float> read(const string filename, int & numDocs, int & numTerms, vector<string> & docTitles, vector<string> & terms) {
	std::ifstream f(filename);
	assert(f.is_open());
	f >> numDocs >> numTerms;
	
	char buf[1024];
	f.getline(buf, 1024);
	for (auto i = 0; i < numDocs; i++) {
		f.getline(buf, 1024);
		docTitles.push_back(string(buf));
	}
	for (auto i = 0; i < numTerms; i++) {
		f.getline(buf, 1024);
		terms.push_back(string(buf));
	}
	auto term_freq_mat = shared_ptr<float>(new float[numDocs * numTerms]);
	memset(term_freq_mat.get(), 0, sizeof(float) * numDocs * numTerms);
	for (auto docIdx = 0; docIdx < numDocs; docIdx++) {
		int numTermsInDoc;
		f >> numTermsInDoc;
		for (auto j = 0; j < numTermsInDoc; j++) {
			int termIdx, freq;
			f >> termIdx >> freq;
			term_freq_mat.get()[docIdx * numTerms + termIdx] = freq;
		}
	}
	f.close();
	return term_freq_mat;
}

void printConcepts(const int numConcept, const int numTerms, const int numDocs, const std::vector<std::vector<std::tuple<int, float>>> topDocs, const std::vector<std::vector<std::tuple<int, float>>> topTerms, const vector<string> terms, const vector<string> docTitles) {
	for (int c = 0; c < numConcept; c++) {
		std::cout << "Concept " << c << ":" << std::endl;
		std::cout << "\t" << "Top Terms: ";
		for (int i = 0; i < numTerms; i++) {
			std::cout << terms[get<0>(topTerms[c][i])] << "(" << get<1>(topTerms[c][i]) << "), ";
		}
		std::cout << std::endl;
		std::cout << "\t" << "Top Docs: ";
		for (int i = 0; i < numDocs; i++) {
			std::cout << "\"" << docTitles[get<0>(topDocs[c][i])] << "\"(" << get<1>(topDocs[c][i]) << "), ";
		}
		std::cout << std::endl;
	}
}

void printTermRelated(const int termIdx, const int numTop, const std::vector<std::tuple<int, float>> termTerm, const std::vector<std::tuple<int, float>> termDoc, const vector<string> terms, const vector<string> docTitles) {
	auto count = 0;
	std::cout << "Top " << numTop << " terms correlates to term index " << termIdx << " :" << std::endl;
	std::cout << "\t";
	count = 0;
	for (auto & t : termTerm) {
		std::cout << terms[std::get<0>(t)] << "(" << std::get<1>(t) << "), ";
		count++;
		if (count >= numTop) {
			break;
		}
	}
	std::cout << std::endl;

	std::cout << "Top " << numTop << " docs correlates to term index " << termIdx << " :" << std::endl;
	std::cout << "\t";
	count = 0;
	for (auto & t : termDoc) {
		std::cout << "\"" << docTitles[std::get<0>(t)] << "\"(" << std::get<1>(t) << "), ";
		count++;
		if (count >= numTop) {
			break;
		}
	}
	std::cout << std::endl;
}

void lsa_demo(const int numDocs, const int numTerms, float * docTermFreq, const vector<string> docTitles, const vector<string> terms, const int numConcepts, const int numShowTop) {
		
	auto m = numDocs;
	auto n = numTerms;
	auto k = numConcepts;
	printMatrix("TF", m, n, docTermFreq);
	tfidf(m, n, docTermFreq);
	printMatrix("IDF", m, n, docTermFreq);
	auto transposed = false;
	if (numDocs < numTerms) {
		transpose(numDocs, numTerms, docTermFreq);
		m = numTerms;
		n = numDocs;
		transposed = true;
	}	
	auto u = unique_ptr<float>(new float[m * k]);
	auto s = unique_ptr<float>(new float[k]);
	auto vt = unique_ptr<float>(new float[k * n]);
	approximate_svd(m, n, k, docTermFreq, u.get(), s.get(), vt.get());
	auto v = unique_ptr<float>();
	v.swap(vt);
	transpose(k, n, v.get());
	if (transposed) {
		u.swap(v);
		auto temp = m;
		m = n;
		n = temp;
	}
	printMatrix("U", m, k, u.get());
	printMatrix("S", 1, k, s.get());
	printMatrix("V", n, k, v.get());
	// print concepts
	auto topTerms = topElementsInTopConcepts(n, k, v.get(), numShowTop, numShowTop);
	auto topDocs = topElementsInTopConcepts(m, k, u.get(), numShowTop, numShowTop);
	printConcepts(numShowTop, numShowTop, numShowTop, topDocs, topTerms, terms, docTitles);

	//corrolated
	multiplyByDiagonalMatrix(n, k, v.get(), s.get());
	printMatrix("VS", n, k, v.get());
	rowsNormalized(n, k, v.get());
	printMatrix("normVS", n, k, v.get());
	multiplyByDiagonalMatrix(m, k, u.get(), s.get());
	printMatrix("US", m, k, u.get());
	rowsNormalized(m, k, u.get());
	printMatrix("normUS", m, k, u.get());
	while (true) {
		try {
			std::cout << std::endl;
			std::cout << "Query term: ";
			char buf[1024];
			std::cin.getline(buf, 1024);
			string term(buf);
			//trim
			if (term.empty()) {
				break;
			}
			term.erase(0, term.find_first_not_of(" "));
			term.erase(term.find_last_not_of(" ") + 1);
			//lower case
			std::transform(term.begin(), term.end(), term.begin(), ::tolower);
			auto termIdx = -1;
			for (auto i = 0; i < numTerms; i++) {
				if (terms[i] == term) {
					termIdx = i;
					break;
				}
			}
			if (termIdx < 0) {
				std::cout << "Term not exist." << std::endl;
				continue;
			}
			auto termTerm = topsForTerm(n, k, v.get(), n, k, v.get(), termIdx);
			auto termDoc = topsForTerm(m, k, u.get(), n, k, v.get(), termIdx);

			printTermRelated(termIdx, numShowTop, termTerm, termDoc, terms, docTitles);
		} catch (std::exception e) {
			std::cout << e.what() << std::endl;
		}
	}

}

int main(int argc, char *argv[]){
	std::cout << "This is CUDA-SPARCE-SMALL LSA demo for EE451 Team7 course project. CUDA-SPARCE-LARGE is on the way. SPARK-SPARCE-LARGE is ready else where." << std::endl;

	assert(argc >= 2);
	int numDocs, numTerms;
	auto docTitles = vector<string>();
	auto terms = vector<string>();
	auto filename = string(argv[1]);
	auto numConcepts = std::numeric_limits<int>::max();
	if (argc >= 3) {
		numConcepts = atoi(argv[2]);
	}
	auto numShowTop = 5;
	if (argc >= 4) {
		numShowTop = atoi(argv[3]);
	}
	cout << "filename=" << filename << "\t" << "#concept=" << numConcepts << "\t" << "#show=" << numShowTop << endl;
	
	auto doc_term_freq_mat = read(filename, numDocs, numTerms, docTitles, terms);
	std::cout << "Read " << numDocs << " docs, " << numTerms << " terms." << std::endl;
	auto minNumConcept = std::min(numDocs, numTerms);
	if (minNumConcept < numConcepts) {
		cout << "Note: #concept " << numConcepts << " -> " << minNumConcept << endl;
		numConcepts = minNumConcept;
	}
	auto minNumShowTop = std::min(numConcepts, numShowTop);
	if (minNumShowTop < numShowTop) {
		cout << "Note: #show " << numShowTop << " -> " << minNumShowTop << endl;
		numShowTop = minNumShowTop;
	}

	lsa_demo(numDocs, numTerms, doc_term_freq_mat.get(), docTitles, terms, numConcepts, numShowTop);
    return 0;
}
